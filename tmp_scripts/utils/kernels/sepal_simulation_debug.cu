
#include <hip/hip_runtime.h>
extern "C" {
    
    __device__ float compute_entropy_device(
        const float* __restrict__ conc,
        const int* __restrict__ sat,             // INDEX ARRAY: saturated node indices
        int n_sat
    ) {
        float total_sum = 0.0f;
        float entropy = 0.0f;
        
        // Compute total sum of saturated concentrations
        for (int i = 0; i < n_sat; i++) {
            float val = conc[sat[i]];  // sat[i] is INDEX into conc array
            if (val > 0.0f) {
                total_sum += val;
            }
        }
        
        if (total_sum <= 0.0f) {
            return 0.0f;
        }
        
        // Compute entropy
        for (int i = 0; i < n_sat; i++) {
            float val = conc[sat[i]];  // sat[i] is INDEX into conc array
            if (val > 0.0f) {
                float normalized = val / total_sum;
                entropy += -normalized * logf(normalized);
            }
        }
        
        return entropy / n_sat;
    }

    __global__ void sepal_simulation_debug(
        const float* __restrict__ gene_data,
        const int* __restrict__ sat,
        const int* __restrict__ sat_idx,
        const int* __restrict__ unsat,
        const int* __restrict__ unsat_idx,
        float* __restrict__ result,
        int n_cells, int n_sat, int n_unsat, int sat_thresh,
        int max_neighs, int n_iter, float dt, float thresh
    )
    {
        int tid = threadIdx.x;
        int blockSize = blockDim.x;
        
        // FIXED: Add proper memory layout like the main kernel
        extern __shared__ float shared_mem[];
        
        // Calculate offsets explicitly to prevent overlap
        const int conc_offset = 0;
        const int nhood_offset = n_cells;  
        const int dcdt_offset = n_cells + n_sat;
        
        float* concentration = &shared_mem[conc_offset];     // [n_cells]
        float* neighbor_sums = &shared_mem[nhood_offset];    // [n_sat]
        float* derivatives = &shared_mem[dcdt_offset];       // [n_cells]
        
        __shared__ bool has_converged;
        __shared__ int convergence_iteration;
        
        if (tid == 0) {
            has_converged = false;
            convergence_iteration = n_iter;
            
            printf("=== GPU DEBUG ===\n");
            printf("n_cells: %d, n_sat: %d, n_unsat: %d\n", n_cells, n_sat, n_unsat);
            printf("Memory layout: conc[0:%d] nhood[%d:%d] dcdt[%d:%d]\n", 
                   n_cells-1, nhood_offset, nhood_offset+n_sat-1, 
                   dcdt_offset, dcdt_offset+n_cells-1);
        }
        
        // Initialize
        for (int i = tid; i < n_cells; i += blockSize) {
            concentration[i] = gene_data[i];
        }
        for (int i = tid; i < n_sat; i += blockSize) {
            neighbor_sums[i] = 0.0f;
        }
        for (int i = tid; i < n_cells; i += blockSize) {
            derivatives[i] = 0.0f;
        }
        
        __syncthreads();
        
        // Debug initial state
        if (tid == 0) {
            printf("Initial conc[0:5]: %.6f %.6f %.6f %.6f %.6f\n", 
                   concentration[0], concentration[1], concentration[2], 
                   concentration[3], concentration[4]);
            printf("sat[0:3]: %d %d %d\n", sat[0], sat[1], sat[2]);
            if (n_unsat > 0) {
                printf("unsat[0:3]: %d %d %d\n", unsat[0], 
                       n_unsat > 1 ? unsat[1] : -1, 
                       n_unsat > 2 ? unsat[2] : -1);
            }
        }
        
        float prev_entropy = 1.0f;
        const float D = 1.0f;
        const float h = 1.0f;
        
        // Main simulation loop - debug first 5 iterations
        for (int iter = 0; iter < min(5, n_iter); iter++) {
            
            // Phase 1: Neighborhood sums
            for (int i = tid; i < n_sat; i += blockSize) {
                float sum = 0.0f;
                for (int j = 0; j < sat_thresh; j++) {
                    int neighbor_idx = sat_idx[i * sat_thresh + j];
                    if (neighbor_idx >= 0 && neighbor_idx < n_cells) {  // BOUNDS CHECK
                        sum += concentration[neighbor_idx];
                    }
                }
                neighbor_sums[i] = sum;
            }
            
            __syncthreads();
            
            // Debug neighborhood sums
            if (tid == 0) {
                printf("Iter %d:\n", iter);
                printf("  nhood[0:3]: %.6f %.6f %.6f\n", 
                       neighbor_sums[0], neighbor_sums[1], neighbor_sums[2]);
                printf("  conc[sat][0:3]: %.6f %.6f %.6f\n",
                       concentration[sat[0]], concentration[sat[1]], concentration[sat[2]]);
            }
            
            // Phase 2: Derivatives
            for (int i = tid; i < n_sat; i += blockSize) {
                int sat_node_idx = sat[i];
                if (sat_node_idx >= 0 && sat_node_idx < n_cells) {  // BOUNDS CHECK
                    float center = concentration[sat_node_idx];
                    float neighbors = neighbor_sums[i];
                    float d2;
                    
                    if (max_neighs == 4) {
                        d2 = (neighbors - 4.0f * center) / (h * h);
                    } else if (max_neighs == 6) {
                        d2 = (2.0f * (neighbors - 6.0f * center) / (3.0f * h * h));
                    }
                    
                    derivatives[sat_node_idx] = D * d2;
                }
            }
            
            __syncthreads();
            
            // Debug derivatives
            if (tid == 0) {
                printf("  d2[0:3]: %.6f %.6f %.6f\n",
                       derivatives[sat[0]], derivatives[sat[1]], derivatives[sat[2]]);
            }
            
            // Phase 3: Update saturated
            for (int i = tid; i < n_sat; i += blockSize) {
                int sat_node_idx = sat[i];
                if (sat_node_idx >= 0 && sat_node_idx < n_cells) {  // BOUNDS CHECK
                    concentration[sat_node_idx] += derivatives[sat_node_idx] * dt;
                    concentration[sat_node_idx] = fmaxf(0.0f, concentration[sat_node_idx]);
                }
            }
            
            // Phase 4: Update unsaturated
            for (int i = tid; i < n_unsat; i += blockSize) {
                int unsat_node_idx = unsat[i];
                int mapped_sat_idx = unsat_idx[i];
                if (unsat_node_idx >= 0 && unsat_node_idx < n_cells && 
                    mapped_sat_idx >= 0 && mapped_sat_idx < n_cells) {  // BOUNDS CHECK
                    concentration[unsat_node_idx] += derivatives[mapped_sat_idx] * dt;
                    concentration[unsat_node_idx] = fmaxf(0.0f, concentration[unsat_node_idx]);
                }
            }
            
            __syncthreads();
            
            // Debug updated concentrations
            if (tid == 0) {
                printf("  After update conc[sat][0:3]: %.6f %.6f %.6f\n",
                       concentration[sat[0]], concentration[sat[1]], concentration[sat[2]]);
                if (n_unsat > 0) {
                    printf("  After update conc[unsat][0:3]: %.6f %.6f %.6f\n",
                           concentration[unsat[0]], 
                           n_unsat > 1 ? concentration[unsat[1]] : 0.0f,
                           n_unsat > 2 ? concentration[unsat[2]] : 0.0f);
                }
            }
            
            // Phase 5: Check convergence
            if (tid == 0) {
                float entropy = compute_entropy_device(concentration, sat, n_sat);
                float entropy_diff = fabsf(entropy - prev_entropy);
                
                printf("  entropy: %.6f, diff: %.6f\n", entropy, entropy_diff);
                
                if (entropy_diff <= thresh) {
                    has_converged = true;
                    convergence_iteration = iter;
                    printf("  GPU CONVERGED at iteration %d\n", iter);
                }
                
                prev_entropy = entropy;
            }
            
            __syncthreads();
            
            if (has_converged) {
                break;
            }
        }
        
        // Continue without debug prints for remaining iterations  
        for (int iter = 5; iter < n_iter; iter++) {
            // Phase 1: Neighborhood sums
            for (int i = tid; i < n_sat; i += blockSize) {
                float sum = 0.0f;
                for (int j = 0; j < sat_thresh; j++) {
                    int neighbor_idx = sat_idx[i * sat_thresh + j];
                    if (neighbor_idx >= 0 && neighbor_idx < n_cells) {
                        sum += concentration[neighbor_idx];
                    }
                }
                neighbor_sums[i] = sum;
            }
            
            __syncthreads();
            
            // Phase 2: Derivatives
            for (int i = tid; i < n_sat; i += blockSize) {
                int sat_node_idx = sat[i];
                if (sat_node_idx >= 0 && sat_node_idx < n_cells) {
                    float center = concentration[sat_node_idx];
                    float neighbors = neighbor_sums[i];
                    float d2;
                    
                    if (max_neighs == 4) {
                        d2 = (neighbors - 4.0f * center) / (h * h);
                    } else if (max_neighs == 6) {
                        d2 = (2.0f * (neighbors - 6.0f * center) / (3.0f * h * h));
                    }
                    
                    derivatives[sat_node_idx] = D * d2;
                }
            }
            
            __syncthreads();
            
            // Phase 3: Update saturated
            for (int i = tid; i < n_sat; i += blockSize) {
                int sat_node_idx = sat[i];
                if (sat_node_idx >= 0 && sat_node_idx < n_cells) {
                    concentration[sat_node_idx] += derivatives[sat_node_idx] * dt;
                    concentration[sat_node_idx] = fmaxf(0.0f, concentration[sat_node_idx]);
                }
            }
            
            // Phase 4: Update unsaturated
            for (int i = tid; i < n_unsat; i += blockSize) {
                int unsat_node_idx = unsat[i];
                int mapped_sat_idx = unsat_idx[i];
                if (unsat_node_idx >= 0 && unsat_node_idx < n_cells && 
                    mapped_sat_idx >= 0 && mapped_sat_idx < n_cells) {
                    concentration[unsat_node_idx] += derivatives[mapped_sat_idx] * dt;
                    concentration[unsat_node_idx] = fmaxf(0.0f, concentration[unsat_node_idx]);
                }
            }
            
            __syncthreads();
            
            // Phase 5: Check convergence
            if (tid == 0) {
                float entropy = compute_entropy_device(concentration, sat, n_sat);
                float entropy_diff = fabsf(entropy - prev_entropy);
                
                if (entropy_diff <= thresh) {
                    has_converged = true;
                    convergence_iteration = iter;
                }
                
                prev_entropy = entropy;
            }
            
            __syncthreads();
            
            if (has_converged) {
                break;
            }
        }
        
        // Write result
        if (tid == 0) {
            if (has_converged) {
                result[0] = dt * convergence_iteration;
                printf("GPU Final result: %.6f\n", result[0]);
            } else {
                result[0] = -999999.0f;
                printf("GPU No convergence\n");
            }
        }
    }
}