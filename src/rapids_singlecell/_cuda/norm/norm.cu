#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_norm.cuh"

namespace nb = nanobind;

template <typename T>
static inline void launch_dense_row_scale(std::uintptr_t data_ptr, int nrows, int ncols,
                                          T target_sum) {
  dim3 block(128);
  dim3 grid((nrows + block.x - 1) / block.x);
  T* data = reinterpret_cast<T*>(data_ptr);
  dense_row_scale_kernel<T><<<grid, block>>>(data, nrows, ncols, target_sum);
}

template <typename T>
static inline void launch_csr_row_scale(std::uintptr_t indptr_ptr, std::uintptr_t data_ptr,
                                        int nrows, T target_sum) {
  dim3 block(128);
  dim3 grid((nrows + block.x - 1) / block.x);
  const int* indptr = reinterpret_cast<const int*>(indptr_ptr);
  T* data = reinterpret_cast<T*>(data_ptr);
  csr_row_scale_kernel<T><<<grid, block>>>(indptr, data, nrows, target_sum);
}

template <typename T>
static inline void launch_csr_sum_major(std::uintptr_t indptr_ptr, std::uintptr_t data_ptr,
                                        std::uintptr_t sums_ptr, int major) {
  dim3 block(64);
  dim3 grid(major);
  std::size_t smem = static_cast<std::size_t>(block.x) * sizeof(T);
  const int* indptr = reinterpret_cast<const int*>(indptr_ptr);
  const T* data = reinterpret_cast<const T*>(data_ptr);
  T* sums = reinterpret_cast<T*>(sums_ptr);
  csr_sum_major_kernel<T><<<grid, block, smem>>>(indptr, data, sums, major);
}

NB_MODULE(_norm_cuda, m) {
  m.def("mul_dense",
        [](std::uintptr_t data, int nrows, int ncols, double target_sum, int itemsize) {
          if (itemsize == 4) {
            launch_dense_row_scale<float>(data, nrows, ncols, (float)target_sum);
          } else if (itemsize == 8) {
            launch_dense_row_scale<double>(data, nrows, ncols, target_sum);
          } else {
            throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
          }
        });

  m.def("mul_csr",
        [](std::uintptr_t indptr, std::uintptr_t data, int nrows, double target_sum, int itemsize) {
          if (itemsize == 4) {
            launch_csr_row_scale<float>(indptr, data, nrows, (float)target_sum);
          } else if (itemsize == 8) {
            launch_csr_row_scale<double>(indptr, data, nrows, target_sum);
          } else {
            throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
          }
        });

  m.def("sum_major", [](std::uintptr_t indptr, std::uintptr_t data, std::uintptr_t sums, int major,
                        int itemsize) {
    if (itemsize == 4) {
      launch_csr_sum_major<float>(indptr, data, sums, major);
    } else if (itemsize == 8) {
      launch_csr_sum_major<double>(indptr, data, sums, major);
    } else {
      throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
    }
  });
}
