#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_autocorr.cuh"

namespace nb = nanobind;
using namespace nb::literals;

static inline void launch_morans_dense(std::uintptr_t data_centered, std::uintptr_t adj_row_ptr,
                                       std::uintptr_t adj_col_ind, std::uintptr_t adj_data,
                                       std::uintptr_t num, int n_samples, int n_features,
                                       hipStream_t stream) {
  dim3 block(8, 8);
  dim3 grid((n_features + block.x - 1) / block.x, (n_samples + block.y - 1) / block.y);
  morans_I_num_dense_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const float*>(data_centered), reinterpret_cast<const int*>(adj_row_ptr),
      reinterpret_cast<const int*>(adj_col_ind), reinterpret_cast<const float*>(adj_data),
      reinterpret_cast<float*>(num), n_samples, n_features);
}

static inline void launch_morans_sparse(std::uintptr_t adj_row_ptr, std::uintptr_t adj_col_ind,
                                        std::uintptr_t adj_data, std::uintptr_t data_row_ptr,
                                        std::uintptr_t data_col_ind, std::uintptr_t data_values,
                                        int n_samples, int n_features, std::uintptr_t mean_array,
                                        std::uintptr_t num, hipStream_t stream) {
  dim3 block(1024);
  dim3 grid(n_samples);
  morans_I_num_sparse_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(adj_row_ptr), reinterpret_cast<const int*>(adj_col_ind),
      reinterpret_cast<const float*>(adj_data), reinterpret_cast<const int*>(data_row_ptr),
      reinterpret_cast<const int*>(data_col_ind), reinterpret_cast<const float*>(data_values),
      n_samples, n_features, reinterpret_cast<const float*>(mean_array),
      reinterpret_cast<float*>(num));
}

static inline void launch_gearys_dense(std::uintptr_t data, std::uintptr_t adj_row_ptr,
                                       std::uintptr_t adj_col_ind, std::uintptr_t adj_data,
                                       std::uintptr_t num, int n_samples, int n_features,
                                       hipStream_t stream) {
  dim3 block(8, 8);
  dim3 grid((n_features + block.x - 1) / block.x, (n_samples + block.y - 1) / block.y);
  gearys_C_num_dense_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const float*>(data), reinterpret_cast<const int*>(adj_row_ptr),
      reinterpret_cast<const int*>(adj_col_ind), reinterpret_cast<const float*>(adj_data),
      reinterpret_cast<float*>(num), n_samples, n_features);
}

static inline void launch_gearys_sparse(std::uintptr_t adj_row_ptr, std::uintptr_t adj_col_ind,
                                        std::uintptr_t adj_data, std::uintptr_t data_row_ptr,
                                        std::uintptr_t data_col_ind, std::uintptr_t data_values,
                                        int n_samples, int n_features, std::uintptr_t num,
                                        hipStream_t stream) {
  dim3 block(1024);
  dim3 grid(n_samples);
  gearys_C_num_sparse_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(adj_row_ptr), reinterpret_cast<const int*>(adj_col_ind),
      reinterpret_cast<const float*>(adj_data), reinterpret_cast<const int*>(data_row_ptr),
      reinterpret_cast<const int*>(data_col_ind), reinterpret_cast<const float*>(data_values),
      n_samples, n_features, reinterpret_cast<float*>(num));
}

static inline void launch_pre_den_sparse(std::uintptr_t data_col_ind, std::uintptr_t data_values,
                                         int nnz, std::uintptr_t mean_array, std::uintptr_t den,
                                         std::uintptr_t counter, hipStream_t stream) {
  dim3 block(32);
  dim3 grid((nnz + block.x - 1) / block.x);
  pre_den_sparse_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(data_col_ind), reinterpret_cast<const float*>(data_values), nnz,
      reinterpret_cast<const float*>(mean_array), reinterpret_cast<float*>(den),
      reinterpret_cast<int*>(counter));
}

NB_MODULE(_autocorr_cuda, m) {
  m.def(
      "morans_dense",
      [](std::uintptr_t data_centered, std::uintptr_t adj_row_ptr, std::uintptr_t adj_col_ind,
         std::uintptr_t adj_data, std::uintptr_t num, int n_samples, int n_features,
         std::uintptr_t stream) {
        launch_morans_dense(data_centered, adj_row_ptr, adj_col_ind, adj_data, num, n_samples,
                            n_features, (hipStream_t)stream);
      },
      "data_centered"_a, nb::kw_only(), "adj_row_ptr"_a, "adj_col_ind"_a, "adj_data"_a, "num"_a,
      "n_samples"_a, "n_features"_a, "stream"_a = 0);
  m.def(
      "morans_sparse",
      [](std::uintptr_t adj_row_ptr, std::uintptr_t adj_col_ind, std::uintptr_t adj_data,
         std::uintptr_t data_row_ptr, std::uintptr_t data_col_ind, std::uintptr_t data_values,
         int n_samples, int n_features, std::uintptr_t mean_array, std::uintptr_t num,
         std::uintptr_t stream) {
        launch_morans_sparse(adj_row_ptr, adj_col_ind, adj_data, data_row_ptr, data_col_ind,
                             data_values, n_samples, n_features, mean_array, num,
                             (hipStream_t)stream);
      },
      "adj_row_ptr"_a, "adj_col_ind"_a, "adj_data"_a, nb::kw_only(), "data_row_ptr"_a,
      "data_col_ind"_a, "data_values"_a, "n_samples"_a, "n_features"_a, "mean_array"_a, "num"_a,
      "stream"_a = 0);
  m.def(
      "gearys_dense",
      [](std::uintptr_t data, std::uintptr_t adj_row_ptr, std::uintptr_t adj_col_ind,
         std::uintptr_t adj_data, std::uintptr_t num, int n_samples, int n_features,
         std::uintptr_t stream) {
        launch_gearys_dense(data, adj_row_ptr, adj_col_ind, adj_data, num, n_samples, n_features,
                            (hipStream_t)stream);
      },
      "data"_a, nb::kw_only(), "adj_row_ptr"_a, "adj_col_ind"_a, "adj_data"_a, "num"_a,
      "n_samples"_a, "n_features"_a, "stream"_a = 0);
  m.def(
      "gearys_sparse",
      [](std::uintptr_t adj_row_ptr, std::uintptr_t adj_col_ind, std::uintptr_t adj_data,
         std::uintptr_t data_row_ptr, std::uintptr_t data_col_ind, std::uintptr_t data_values,
         int n_samples, int n_features, std::uintptr_t num, std::uintptr_t stream) {
        launch_gearys_sparse(adj_row_ptr, adj_col_ind, adj_data, data_row_ptr, data_col_ind,
                             data_values, n_samples, n_features, num, (hipStream_t)stream);
      },
      "adj_row_ptr"_a, "adj_col_ind"_a, "adj_data"_a, nb::kw_only(), "data_row_ptr"_a,
      "data_col_ind"_a, "data_values"_a, "n_samples"_a, "n_features"_a, "num"_a, "stream"_a = 0);
  m.def(
      "pre_den_sparse",
      [](std::uintptr_t data_col_ind, std::uintptr_t data_values, int nnz,
         std::uintptr_t mean_array, std::uintptr_t den, std::uintptr_t counter,
         std::uintptr_t stream) {
        launch_pre_den_sparse(data_col_ind, data_values, nnz, mean_array, den, counter,
                              (hipStream_t)stream);
      },
      "data_col_ind"_a, "data_values"_a, nb::kw_only(), "nnz"_a, "mean_array"_a, "den"_a,
      "counter"_a, "stream"_a = 0);
}
