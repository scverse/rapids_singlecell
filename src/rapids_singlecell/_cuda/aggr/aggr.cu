#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

namespace nb = nanobind;

#include "kernels_aggr.cuh"

// Launchers
template <typename T>
static inline void launch_csr_aggr(std::uintptr_t indptr, std::uintptr_t index, std::uintptr_t data,
                                   std::uintptr_t out, std::uintptr_t cats, std::uintptr_t mask,
                                   std::size_t n_cells, std::size_t n_genes, std::size_t n_groups) {
  dim3 grid((unsigned)n_cells);
  dim3 block(64);
  csr_aggr_kernel<T>
      <<<grid, block>>>(reinterpret_cast<const int*>(indptr), reinterpret_cast<const int*>(index),
                        reinterpret_cast<const T*>(data), reinterpret_cast<double*>(out),
                        reinterpret_cast<const int*>(cats), reinterpret_cast<const bool*>(mask),
                        n_cells, n_genes, n_groups);
}

template <typename T>
static inline void launch_csc_aggr(std::uintptr_t indptr, std::uintptr_t index, std::uintptr_t data,
                                   std::uintptr_t out, std::uintptr_t cats, std::uintptr_t mask,
                                   std::size_t n_cells, std::size_t n_genes, std::size_t n_groups) {
  dim3 grid((unsigned)n_genes);
  dim3 block(64);
  csc_aggr_kernel<T>
      <<<grid, block>>>(reinterpret_cast<const int*>(indptr), reinterpret_cast<const int*>(index),
                        reinterpret_cast<const T*>(data), reinterpret_cast<double*>(out),
                        reinterpret_cast<const int*>(cats), reinterpret_cast<const bool*>(mask),
                        n_cells, n_genes, n_groups);
}

template <typename T>
static inline void launch_csr_to_coo(std::uintptr_t indptr, std::uintptr_t index,
                                     std::uintptr_t data, std::uintptr_t row, std::uintptr_t col,
                                     std::uintptr_t ndata, std::uintptr_t cats, std::uintptr_t mask,
                                     int n_cells) {
  dim3 grid((unsigned)n_cells);
  dim3 block(64);
  csr_to_coo_kernel<T><<<grid, block>>>(
      reinterpret_cast<const int*>(indptr), reinterpret_cast<const int*>(index),
      reinterpret_cast<const T*>(data), reinterpret_cast<int*>(row), reinterpret_cast<int*>(col),
      reinterpret_cast<double*>(ndata), reinterpret_cast<const int*>(cats),
      reinterpret_cast<const bool*>(mask), n_cells);
}

template <typename T>
static inline void launch_dense_C(std::uintptr_t data, std::uintptr_t out, std::uintptr_t cats,
                                  std::uintptr_t mask, std::size_t n_cells, std::size_t n_genes,
                                  std::size_t n_groups) {
  dim3 block(256);
  dim3 grid((unsigned)((n_cells * n_genes + block.x - 1) / block.x));
  dense_aggr_kernel_C<T>
      <<<grid, block>>>(reinterpret_cast<const T*>(data), reinterpret_cast<double*>(out),
                        reinterpret_cast<const int*>(cats), reinterpret_cast<const bool*>(mask),
                        n_cells, n_genes, n_groups);
}

template <typename T>
static inline void launch_dense_F(std::uintptr_t data, std::uintptr_t out, std::uintptr_t cats,
                                  std::uintptr_t mask, std::size_t n_cells, std::size_t n_genes,
                                  std::size_t n_groups) {
  dim3 block(256);
  dim3 grid((unsigned)((n_cells * n_genes + block.x - 1) / block.x));
  dense_aggr_kernel_F<T>
      <<<grid, block>>>(reinterpret_cast<const T*>(data), reinterpret_cast<double*>(out),
                        reinterpret_cast<const int*>(cats), reinterpret_cast<const bool*>(mask),
                        n_cells, n_genes, n_groups);
}

// Unified dispatchers
static inline void sparse_aggr_dispatch(std::uintptr_t indptr, std::uintptr_t index,
                                        std::uintptr_t data, std::uintptr_t out,
                                        std::uintptr_t cats, std::uintptr_t mask,
                                        std::size_t n_cells, std::size_t n_genes,
                                        std::size_t n_groups, bool is_csc, int dtype_itemsize) {
  if (is_csc) {
    if (dtype_itemsize == 4) {
      launch_csc_aggr<float>(indptr, index, data, out, cats, mask, n_cells, n_genes, n_groups);
    } else {
      launch_csc_aggr<double>(indptr, index, data, out, cats, mask, n_cells, n_genes, n_groups);
    }
  } else {
    if (dtype_itemsize == 4) {
      launch_csr_aggr<float>(indptr, index, data, out, cats, mask, n_cells, n_genes, n_groups);
    } else {
      launch_csr_aggr<double>(indptr, index, data, out, cats, mask, n_cells, n_genes, n_groups);
    }
  }
}

static inline void dense_aggr_dispatch(std::uintptr_t data, std::uintptr_t out, std::uintptr_t cats,
                                       std::uintptr_t mask, std::size_t n_cells,
                                       std::size_t n_genes, std::size_t n_groups, bool is_fortran,
                                       int dtype_itemsize) {
  if (is_fortran) {
    if (dtype_itemsize == 4) {
      launch_dense_F<float>(data, out, cats, mask, n_cells, n_genes, n_groups);
    } else {
      launch_dense_F<double>(data, out, cats, mask, n_cells, n_genes, n_groups);
    }
  } else {
    if (dtype_itemsize == 4) {
      launch_dense_C<float>(data, out, cats, mask, n_cells, n_genes, n_groups);
    } else {
      launch_dense_C<double>(data, out, cats, mask, n_cells, n_genes, n_groups);
    }
  }
}

static inline void csr_to_coo_dispatch(std::uintptr_t indptr, std::uintptr_t index,
                                       std::uintptr_t data, std::uintptr_t row, std::uintptr_t col,
                                       std::uintptr_t ndata, std::uintptr_t cats,
                                       std::uintptr_t mask, int n_cells, int dtype_itemsize) {
  if (dtype_itemsize == 4) {
    launch_csr_to_coo<float>(indptr, index, data, row, col, ndata, cats, mask, n_cells);
  } else {
    launch_csr_to_coo<double>(indptr, index, data, row, col, ndata, cats, mask, n_cells);
  }
}

// variance launcher
static inline void launch_sparse_var(std::uintptr_t indptr, std::uintptr_t index,
                                     std::uintptr_t data, std::uintptr_t mean_data,
                                     std::uintptr_t n_cells, int dof, int n_groups) {
  dim3 grid((unsigned)n_groups);
  dim3 block(64);
  sparse_var_kernel<<<grid, block>>>(
      reinterpret_cast<const int*>(indptr), reinterpret_cast<const int*>(index),
      reinterpret_cast<double*>(data), reinterpret_cast<const double*>(mean_data),
      reinterpret_cast<double*>(n_cells), dof, n_groups);
}

NB_MODULE(_aggr_cuda, m) {
  m.def("sparse_aggr", &sparse_aggr_dispatch);
  m.def("dense_aggr", &dense_aggr_dispatch);
  m.def("csr_to_coo", &csr_to_coo_dispatch);
  m.def("sparse_var", &launch_sparse_var);
}
