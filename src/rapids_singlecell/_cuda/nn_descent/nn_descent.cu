#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_dist.cuh"

namespace nb = nanobind;
using namespace nb::literals;

static inline void launch_sqeuclidean(std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs,
                                      long long n_samples, long long n_features,
                                      long long n_neighbors, hipStream_t stream) {
  dim3 block(32);
  dim3 grid((unsigned)((n_samples + block.x - 1) / block.x));
  compute_distances_sqeuclidean_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const float*>(data), reinterpret_cast<float*>(out),
      reinterpret_cast<const unsigned int*>(pairs), n_samples, n_features, n_neighbors);
}

static inline void launch_cosine(std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs,
                                 long long n_samples, long long n_features, long long n_neighbors,
                                 hipStream_t stream) {
  dim3 block(32);
  dim3 grid((unsigned)((n_samples + block.x - 1) / block.x));
  compute_distances_cosine_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const float*>(data), reinterpret_cast<float*>(out),
      reinterpret_cast<const unsigned int*>(pairs), n_samples, n_features, n_neighbors);
}

static inline void launch_inner(std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs,
                                long long n_samples, long long n_features, long long n_neighbors,
                                hipStream_t stream) {
  dim3 block(32);
  dim3 grid((unsigned)((n_samples + block.x - 1) / block.x));
  compute_distances_inner_kernel<<<grid, block, 0, stream>>>(
      reinterpret_cast<const float*>(data), reinterpret_cast<float*>(out),
      reinterpret_cast<const unsigned int*>(pairs), n_samples, n_features, n_neighbors);
}

NB_MODULE(_nn_descent_cuda, m) {
  m.def(
      "sqeuclidean",
      [](std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs, long long n_samples,
         long long n_features, long long n_neighbors, std::uintptr_t stream) {
        launch_sqeuclidean(data, out, pairs, n_samples, n_features, n_neighbors,
                           (hipStream_t)stream);
      },
      "data"_a, nb::kw_only(), "out"_a, "pairs"_a, "n_samples"_a, "n_features"_a, "n_neighbors"_a,
      "stream"_a = 0);
  m.def(
      "cosine",
      [](std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs, long long n_samples,
         long long n_features, long long n_neighbors, std::uintptr_t stream) {
        launch_cosine(data, out, pairs, n_samples, n_features, n_neighbors, (hipStream_t)stream);
      },
      "data"_a, nb::kw_only(), "out"_a, "pairs"_a, "n_samples"_a, "n_features"_a, "n_neighbors"_a,
      "stream"_a = 0);
  m.def(
      "inner",
      [](std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs, long long n_samples,
         long long n_features, long long n_neighbors, std::uintptr_t stream) {
        launch_inner(data, out, pairs, n_samples, n_features, n_neighbors, (hipStream_t)stream);
      },
      "data"_a, nb::kw_only(), "out"_a, "pairs"_a, "n_samples"_a, "n_features"_a, "n_neighbors"_a,
      "stream"_a = 0);
}
