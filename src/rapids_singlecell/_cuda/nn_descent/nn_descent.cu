#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_dist.cuh"

namespace nb = nanobind;

static inline void launch_sqeuclidean(std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs,
                                      long long n_samples, long long n_features,
                                      long long n_neighbors) {
  dim3 block(32);
  dim3 grid((unsigned)((n_samples + block.x - 1) / block.x));
  compute_distances_sqeuclidean_kernel<<<grid, block>>>(
      reinterpret_cast<const float*>(data), reinterpret_cast<float*>(out),
      reinterpret_cast<const unsigned int*>(pairs), n_samples, n_features, n_neighbors);
}

static inline void launch_cosine(std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs,
                                 long long n_samples, long long n_features, long long n_neighbors) {
  dim3 block(32);
  dim3 grid((unsigned)((n_samples + block.x - 1) / block.x));
  compute_distances_cosine_kernel<<<grid, block>>>(
      reinterpret_cast<const float*>(data), reinterpret_cast<float*>(out),
      reinterpret_cast<const unsigned int*>(pairs), n_samples, n_features, n_neighbors);
}

static inline void launch_inner(std::uintptr_t data, std::uintptr_t out, std::uintptr_t pairs,
                                long long n_samples, long long n_features, long long n_neighbors) {
  dim3 block(32);
  dim3 grid((unsigned)((n_samples + block.x - 1) / block.x));
  compute_distances_inner_kernel<<<grid, block>>>(
      reinterpret_cast<const float*>(data), reinterpret_cast<float*>(out),
      reinterpret_cast<const unsigned int*>(pairs), n_samples, n_features, n_neighbors);
}

NB_MODULE(_nn_descent_cuda, m) {
  m.def("sqeuclidean", &launch_sqeuclidean);
  m.def("cosine", &launch_cosine);
  m.def("inner", &launch_inner);
}
