#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_scale.cuh"

namespace nb = nanobind;
using namespace nb::literals;

template <typename T>
static inline void launch_csc_scale_diff(std::uintptr_t indptr, std::uintptr_t data,
                                         std::uintptr_t std, int ncols, hipStream_t stream) {
  dim3 block(64);
  dim3 grid(ncols);
  csc_scale_diff_kernel<T><<<grid, block, 0, stream>>>(reinterpret_cast<const int*>(indptr),
                                                       reinterpret_cast<T*>(data),
                                                       reinterpret_cast<const T*>(std), ncols);
}

template <typename T>
static inline void launch_csr_scale_diff(std::uintptr_t indptr, std::uintptr_t indices,
                                         std::uintptr_t data, std::uintptr_t std,
                                         std::uintptr_t mask, T clipper, int nrows,
                                         hipStream_t stream) {
  dim3 block(64);
  dim3 grid(nrows);
  csr_scale_diff_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(indptr), reinterpret_cast<const int*>(indices),
      reinterpret_cast<T*>(data), reinterpret_cast<const T*>(std),
      reinterpret_cast<const int*>(mask), clipper, nrows);
}

template <typename T>
static inline void launch_dense_scale_center_diff(std::uintptr_t data, std::uintptr_t mean,
                                                  std::uintptr_t std, std::uintptr_t mask,
                                                  T clipper, long long nrows, long long ncols,
                                                  hipStream_t stream) {
  dim3 block(32, 32);
  dim3 grid((unsigned)((nrows + block.x - 1) / block.x),
            (unsigned)((ncols + block.y - 1) / block.y));
  dense_scale_center_diff_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<T*>(data), reinterpret_cast<const T*>(mean), reinterpret_cast<const T*>(std),
      reinterpret_cast<const int*>(mask), clipper, nrows, ncols);
}

template <typename T>
static inline void launch_dense_scale_diff(std::uintptr_t data, std::uintptr_t std,
                                           std::uintptr_t mask, T clipper, long long nrows,
                                           long long ncols, hipStream_t stream) {
  dim3 block(32, 32);
  dim3 grid((unsigned)((nrows + block.x - 1) / block.x),
            (unsigned)((ncols + block.y - 1) / block.y));
  dense_scale_diff_kernel<T>
      <<<grid, block, 0, stream>>>(reinterpret_cast<T*>(data), reinterpret_cast<const T*>(std),
                                   reinterpret_cast<const int*>(mask), clipper, nrows, ncols);
}

NB_MODULE(_scale_cuda, m) {
  m.def(
      "csc_scale_diff",
      [](std::uintptr_t indptr, std::uintptr_t data, std::uintptr_t std, int ncols, int itemsize,
         std::uintptr_t stream) {
        if (itemsize == 4)
          launch_csc_scale_diff<float>(indptr, data, std, ncols, (hipStream_t)stream);
        else if (itemsize == 8)
          launch_csc_scale_diff<double>(indptr, data, std, ncols, (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
      },
      "indptr"_a, "data"_a, "std"_a, nb::kw_only(), "ncols"_a, "itemsize"_a, "stream"_a = 0);
  m.def(
      "csr_scale_diff",
      [](std::uintptr_t indptr, std::uintptr_t indices, std::uintptr_t data, std::uintptr_t std,
         std::uintptr_t mask, double clipper, int nrows, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4)
          launch_csr_scale_diff<float>(indptr, indices, data, std, mask, (float)clipper, nrows,
                                       (hipStream_t)stream);
        else if (itemsize == 8)
          launch_csr_scale_diff<double>(indptr, indices, data, std, mask, (double)clipper, nrows,
                                        (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
      },
      "indptr"_a, "indices"_a, "data"_a, "std"_a, "mask"_a, nb::kw_only(), "clipper"_a, "nrows"_a,
      "itemsize"_a, "stream"_a = 0);
  m.def(
      "dense_scale_center_diff",
      [](std::uintptr_t data, std::uintptr_t mean, std::uintptr_t std, std::uintptr_t mask,
         double clipper, long long nrows, long long ncols, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4)
          launch_dense_scale_center_diff<float>(data, mean, std, mask, (float)clipper, nrows, ncols,
                                                (hipStream_t)stream);
        else if (itemsize == 8)
          launch_dense_scale_center_diff<double>(data, mean, std, mask, (double)clipper, nrows,
                                                 ncols, (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
      },
      "data"_a, "mean"_a, "std"_a, "mask"_a, nb::kw_only(), "clipper"_a, "nrows"_a, "ncols"_a,
      "itemsize"_a, "stream"_a = 0);
  m.def(
      "dense_scale_diff",
      [](std::uintptr_t data, std::uintptr_t std, std::uintptr_t mask, double clipper,
         long long nrows, long long ncols, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4)
          launch_dense_scale_diff<float>(data, std, mask, (float)clipper, nrows, ncols,
                                         (hipStream_t)stream);
        else if (itemsize == 8)
          launch_dense_scale_diff<double>(data, std, mask, (double)clipper, nrows, ncols,
                                          (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
      },
      "data"_a, "std"_a, "mask"_a, nb::kw_only(), "clipper"_a, "nrows"_a, "ncols"_a, "itemsize"_a,
      "stream"_a = 0);
}
