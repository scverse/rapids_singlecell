#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_pv.cuh"

namespace nb = nanobind;

static inline void launch_rev_cummin64(std::uintptr_t x, std::uintptr_t y, int n_rows, int m) {
  dim3 block(256);
  dim3 grid((unsigned)((n_rows + block.x - 1) / block.x));
  rev_cummin64_kernel<<<grid, block>>>(reinterpret_cast<const double*>(x),
                                       reinterpret_cast<double*>(y), n_rows, m);
}

NB_MODULE(_pv_cuda, m) {
  m.def("rev_cummin64", [](std::uintptr_t x, std::uintptr_t y, int n_rows, int m) {
    launch_rev_cummin64(x, y, n_rows, m);
  });
}
