#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_nanmean.cuh"

namespace nb = nanobind;

template <typename T>
static inline void launch_nan_mean_minor(std::uintptr_t index, std::uintptr_t data,
                                         std::uintptr_t means, std::uintptr_t nans,
                                         std::uintptr_t mask, int nnz, hipStream_t stream) {
  dim3 block(32);
  dim3 grid((nnz + block.x - 1) / block.x);
  nan_mean_minor_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(index), reinterpret_cast<const T*>(data),
      reinterpret_cast<double*>(means), reinterpret_cast<int*>(nans),
      reinterpret_cast<const bool*>(mask), nnz);
}

template <typename T>
static inline void launch_nan_mean_major(std::uintptr_t indptr, std::uintptr_t index,
                                         std::uintptr_t data, std::uintptr_t means,
                                         std::uintptr_t nans, std::uintptr_t mask, int major,
                                         int minor, hipStream_t stream) {
  dim3 block(64);
  dim3 grid(major);
  nan_mean_major_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(indptr), reinterpret_cast<const int*>(index),
      reinterpret_cast<const T*>(data), reinterpret_cast<double*>(means),
      reinterpret_cast<int*>(nans), reinterpret_cast<const bool*>(mask), major, minor);
}

NB_MODULE(_nanmean_cuda, m) {
  m.def(
      "nan_mean_minor",
      [](std::uintptr_t index, std::uintptr_t data, std::uintptr_t means, std::uintptr_t nans,
         std::uintptr_t mask, int nnz, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4)
          launch_nan_mean_minor<float>(index, data, means, nans, mask, nnz, (hipStream_t)stream);
        else if (itemsize == 8)
          launch_nan_mean_minor<double>(index, data, means, nans, mask, nnz, (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize");
      },
      nb::arg("index"), nb::arg("data"), nb::arg("means"), nb::arg("nans"), nb::arg("mask"),
      nb::arg("nnz"), nb::arg("itemsize"), nb::arg("stream") = 0);

  m.def(
      "nan_mean_major",
      [](std::uintptr_t indptr, std::uintptr_t index, std::uintptr_t data, std::uintptr_t means,
         std::uintptr_t nans, std::uintptr_t mask, int major, int minor, int itemsize,
         std::uintptr_t stream) {
        if (itemsize == 4)
          launch_nan_mean_major<float>(indptr, index, data, means, nans, mask, major, minor,
                                       (hipStream_t)stream);
        else if (itemsize == 8)
          launch_nan_mean_major<double>(indptr, index, data, means, nans, mask, major, minor,
                                        (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize");
      },
      nb::arg("indptr"), nb::arg("index"), nb::arg("data"), nb::arg("means"), nb::arg("nans"),
      nb::arg("mask"), nb::arg("major"), nb::arg("minor"), nb::arg("itemsize"),
      nb::arg("stream") = 0);
}
