#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_qcd.cuh"

namespace nb = nanobind;

template <typename T>
static inline void launch_qc_csr_cells(std::uintptr_t indptr, std::uintptr_t index,
                                       std::uintptr_t data, std::uintptr_t sums_cells,
                                       std::uintptr_t cell_ex, int n_cells, hipStream_t stream) {
  dim3 block(32);
  dim3 grid((n_cells + 31) / 32);
  qc_csr_cells_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(indptr), reinterpret_cast<const int*>(index),
      reinterpret_cast<const T*>(data), reinterpret_cast<T*>(sums_cells),
      reinterpret_cast<int*>(cell_ex), n_cells);
}

template <typename T>
static inline void launch_qc_csr_genes(std::uintptr_t index, std::uintptr_t data,
                                       std::uintptr_t sums_genes, std::uintptr_t gene_ex, int nnz,
                                       hipStream_t stream) {
  int block = 256;
  int grid = (nnz + block - 1) / block;
  qc_csr_genes_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<const int*>(index), reinterpret_cast<const T*>(data),
      reinterpret_cast<T*>(sums_genes), reinterpret_cast<int*>(gene_ex), nnz);
}

template <typename T>
static inline void launch_qc_dense_cells(std::uintptr_t data, std::uintptr_t sums_cells,
                                         std::uintptr_t cell_ex, int n_cells, int n_genes,
                                         hipStream_t stream) {
  dim3 block(16, 16);
  dim3 grid((n_cells + 15) / 16, (n_genes + 15) / 16);
  qc_dense_cells_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<const T*>(data), reinterpret_cast<T*>(sums_cells),
      reinterpret_cast<int*>(cell_ex), n_cells, n_genes);
}

template <typename T>
static inline void launch_qc_dense_genes(std::uintptr_t data, std::uintptr_t sums_genes,
                                         std::uintptr_t gene_ex, int n_cells, int n_genes,
                                         hipStream_t stream) {
  dim3 block(16, 16);
  dim3 grid((n_cells + 15) / 16, (n_genes + 15) / 16);
  qc_dense_genes_kernel<T><<<grid, block, 0, stream>>>(
      reinterpret_cast<const T*>(data), reinterpret_cast<T*>(sums_genes),
      reinterpret_cast<int*>(gene_ex), n_cells, n_genes);
}

NB_MODULE(_qc_dask_cuda, m) {
  m.def(
      "sparse_qc_csr_cells",
      [](std::uintptr_t indptr, std::uintptr_t index, std::uintptr_t data,
         std::uintptr_t sums_cells, std::uintptr_t cell_ex, int n_cells, int itemsize,
         std::uintptr_t stream) {
        if (itemsize == 4)
          launch_qc_csr_cells<float>(indptr, index, data, sums_cells, cell_ex, n_cells,
                                     (hipStream_t)stream);
        else if (itemsize == 8)
          launch_qc_csr_cells<double>(indptr, index, data, sums_cells, cell_ex, n_cells,
                                      (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize");
      },
      nb::arg("indptr"), nb::arg("index"), nb::arg("data"), nb::arg("sums_cells"),
      nb::arg("cell_ex"), nb::arg("n_cells"), nb::arg("itemsize"), nb::arg("stream") = 0);
  m.def(
      "sparse_qc_csr_genes",
      [](std::uintptr_t index, std::uintptr_t data, std::uintptr_t sums_genes,
         std::uintptr_t gene_ex, int nnz, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4)
          launch_qc_csr_genes<float>(index, data, sums_genes, gene_ex, nnz, (hipStream_t)stream);
        else if (itemsize == 8)
          launch_qc_csr_genes<double>(index, data, sums_genes, gene_ex, nnz, (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize");
      },
      nb::arg("index"), nb::arg("data"), nb::arg("sums_genes"), nb::arg("gene_ex"), nb::arg("nnz"),
      nb::arg("itemsize"), nb::arg("stream") = 0);
  m.def(
      "sparse_qc_dense_cells",
      [](std::uintptr_t data, std::uintptr_t sums_cells, std::uintptr_t cell_ex, int n_cells,
         int n_genes, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4)
          launch_qc_dense_cells<float>(data, sums_cells, cell_ex, n_cells, n_genes,
                                       (hipStream_t)stream);
        else if (itemsize == 8)
          launch_qc_dense_cells<double>(data, sums_cells, cell_ex, n_cells, n_genes,
                                        (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize");
      },
      nb::arg("data"), nb::arg("sums_cells"), nb::arg("cell_ex"), nb::arg("n_cells"),
      nb::arg("n_genes"), nb::arg("itemsize"), nb::arg("stream") = 0);
  m.def(
      "sparse_qc_dense_genes",
      [](std::uintptr_t data, std::uintptr_t sums_genes, std::uintptr_t gene_ex, int n_cells,
         int n_genes, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4)
          launch_qc_dense_genes<float>(data, sums_genes, gene_ex, n_cells, n_genes,
                                       (hipStream_t)stream);
        else if (itemsize == 8)
          launch_qc_dense_genes<double>(data, sums_genes, gene_ex, n_cells, n_genes,
                                        (hipStream_t)stream);
        else
          throw nb::value_error("Unsupported itemsize");
      },
      nb::arg("data"), nb::arg("sums_genes"), nb::arg("gene_ex"), nb::arg("n_cells"),
      nb::arg("n_genes"), nb::arg("itemsize"), nb::arg("stream") = 0);
}
