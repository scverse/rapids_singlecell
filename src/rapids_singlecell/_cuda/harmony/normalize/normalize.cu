#include <hip/hip_runtime.h>
#include <nanobind/nanobind.h>
#include <cstdint>

#include "kernels_normalize.cuh"

namespace nb = nanobind;
using namespace nb::literals;

template <typename T>
static inline void launch_normalize(std::uintptr_t X, long long rows, long long cols,
                                    hipStream_t stream) {
  dim3 block(32);
  dim3 grid(rows);
  normalize_kernel_optimized<T><<<grid, block, 0, stream>>>(reinterpret_cast<T*>(X), rows, cols);
}

NB_MODULE(_harmony_normalize_cuda, m) {
  m.def(
      "normalize",
      [](std::uintptr_t X, long long rows, long long cols, int itemsize, std::uintptr_t stream) {
        if (itemsize == 4) {
          launch_normalize<float>(X, rows, cols, (hipStream_t)stream);
        } else if (itemsize == 8) {
          launch_normalize<double>(X, rows, cols, (hipStream_t)stream);
        } else {
          throw nb::value_error("Unsupported itemsize (expected 4 or 8)");
        }
      },
      "X"_a, nb::kw_only(), "rows"_a, "cols"_a, "itemsize"_a, "stream"_a = 0);
}
